#include <stdlib.h>
#include <unistd.h>
#include <random>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
//using namespace std;

//maybe
#include <iostream>
#include <stdio.h>

// I/O parameters used to index argv[]
#define OUTPUT_PATH_ID	1
#define STEPS_ID		2
#define BLOCK_SIZE_X	3
#define BLOCK_SIZE_Y	4
#define MATRIX_SIZE		5

#define STRLEN			256


// Function to save the last iteration matrix

bool saveGrid2Dr(int *M, int d, char *path){
	FILE *f;
	f = fopen(path,"w");

	if (!f)
		return false;

	char str[STRLEN];
	for (int i = 0; i < d; i++){
		for (int j = 0; j < d; j++){
			sprintf(str,"%d ",M[i*d+j]);
			fprintf(f,"%s ", str);
		}
		fprintf(f,"\n");
	}
	fclose(f);

	return true;
}


// Kernel for periodic boundary conditions
__device__ int getToroidal(int i, int size){
	if(i < 0){
		return i+size;
	}else{
		if(i > size-1){
			return i-size;
		}
	}
	return i;
}

// Initialize seeds for threads
__global__ void setup_kernel(hiprandState *state, int d){
	int idx =blockDim.x * blockIdx.x + threadIdx.x;
	int idy =blockDim.y * blockIdx.y + threadIdx.y;
	int id = idy*d+idx;
	hiprand_init(1234, id, 0, &state[id]);
}


__global__ void transition_function(int d, int total_steps, int *read_matrix, int *write_matrix, hiprandState *seedStates){
	int x = (blockDim.x*blockIdx.x + threadIdx.x);
	int y = (blockDim.y*blockIdx.y + threadIdx.y);

	int sum;

	if (x < d && y < d){	
		switch(read_matrix[y*d+x]){ 
			case 0: 
				write_matrix[y*d+x] = 0; 
				break;
			
			case 1:
				sum = 0;
				for (int i = -1; i <= 1; i++){
					for (int j = -1; j <= 1; j++){
						if (!(i == 0 && j == 0)){
							int indexi = getToroidal(y+i,d);
							int indexj = getToroidal(x+j,d);
							if (read_matrix[indexi*d+indexj] == 2) 
								sum += 1;
						}
					}
				}

				if (sum > 0){
					float prob = 0.2/7.0*sum + 5.4/7.0;
					hiprandState localState = seedStates[y*d+x];
					float uniformVal = hiprand_uniform(&localState);
					seedStates[y*d+x] = localState;
					int bin_num = 0;
					if (uniformVal > 1-prob)
						bin_num = 1;
					//int bin_num = (uniformVal < 1 - prob)? 0 : 1; // uniform to binomial
					write_matrix[y*d+x] = bin_num + 1;
				}
				else 
					write_matrix[y*d+x] = 1;
				break;	
			case 2:
				write_matrix[y*d+x] = 3;
				break;
			case 3:
				write_matrix[y*d+x] = 3;
				break;
	
	}
		}
}

__global__ void swap(int d, int *read_matrix, int *write_matrix){
	int x = (blockDim.x*blockIdx.x + threadIdx.x);
	int y = (blockDim.y*blockIdx.y + threadIdx.y);
	if (x < d && y < d){
		read_matrix[y*d+x] = write_matrix[y*d+x];
	}
}

void random_seed_matrix(int total_steps, int d, int *seed_array){
	for (int i = 0; i< total_steps*d*d; i++)
		seed_array[i] = rand()%100;
}

void initForest(int d, int *read_matrix, int *write_matrix){
// This function generates the forest (grid) and assigns each cell one of the two possible states: rock (not burnable) or tree (burnable)
	for (int y = 0; y < d; ++y) {
		for (int x = 0; x < d; ++x) {
			int state = rand()%2; 
			read_matrix[y*d+x]=state;
			write_matrix[y*d+x]=state;
		}
	}
	int index_middle = d/2 * d + d/2;
	// introduce a burning cell
	read_matrix[index_middle] = 2;
	write_matrix[index_middle] = 2;
}


//---------------------------------------------------------//
//---------------		MAIN FUNCTION ---------------------//
//---------------------------------------------------------//

int main(int argc, char **argv) {
	srand(1);
	
	// Allocate CPU memory
	int d = atoi(argv[MATRIX_SIZE]);
	int size = d * d * sizeof(int);	
	int total_steps = atoi(argv[STEPS_ID]);

	printf("Dimensio: %d",d);

	int *read_matrix, *write_matrix; 
	read_matrix = (int *)malloc(size);
	write_matrix = (int *)malloc(size);	

	// Block size and number of blocks
	int bs_x, bs_y;
	bs_x = atoi(argv[BLOCK_SIZE_X]);
	bs_y = atoi(argv[BLOCK_SIZE_Y]);


	dim3 block_size(bs_x, bs_y, 1);
	dim3 block_number(ceil((d)/(float)block_size.x), ceil((d)/(float)block_size.y),1);
	
	printf("Files: %d, columnes: %d\n",d,d);
	printf("blocksize_x: %d, blocksize_y: %d\n",bs_x, bs_y);
	printf("Number of blocks (x): %d, Number of blocks (y): %d \n",block_number.x, block_number.y);
	printf("Number of steps: %d",total_steps);

	// Setup seeds
	hiprandState *seedStates;
	hipMalloc((void**) &seedStates, d*d*sizeof(hiprandState));
	setup_kernel<<<block_number,block_size>>>(seedStates,d);

	// Fill read_matrix with initial conditions	
	initForest(d, read_matrix, write_matrix);
	
	// Allocate memory in GPU and copy data  
	int *d_read_matrix, *d_write_matrix;
	
	hipMalloc((void**) &d_read_matrix, size);
	hipMalloc((void**) &d_write_matrix, size);

	hipMemcpy(d_read_matrix, read_matrix, size, hipMemcpyHostToDevice);
	hipMemcpy(d_write_matrix, write_matrix, size, hipMemcpyHostToDevice);

	// Simulation 
	for (int timestep = 0; timestep < total_steps; timestep++){
		// Apply transition function
		transition_function<<<block_number, block_size>>>(d, total_steps, d_read_matrix, d_write_matrix, seedStates);

		hipError_t err = hipGetLastError();
		if (err != hipSuccess){
			printf("CUDA Error in transition_function(): %s\n",hipGetErrorString(err));
		}
		// Swap read and write matrix
		swap<<<block_number, block_size>>>(d, d_read_matrix, d_write_matrix);	
		err = hipGetLastError();
		if (err != hipSuccess){
			printf("CUDA Error in swap(): %s\n",hipGetErrorString(err));
		}
	}

	printf("Saving data...");
	// Copy data from GPU to CPU
	hipMemcpy(read_matrix, d_read_matrix, size, hipMemcpyDeviceToHost);
	hipMemcpy(write_matrix, d_write_matrix, size, hipMemcpyDeviceToHost);

	
	// Copy data to file
	saveGrid2Dr(write_matrix,d,argv[OUTPUT_PATH_ID]);
	
	printf("Releasing memory...\n");
	delete [] read_matrix;
	delete [] write_matrix;
	hipFree(seedStates);
	hipFree(d_read_matrix);
	hipFree(d_write_matrix);
	return 0;
}
